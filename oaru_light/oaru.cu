/*
 * Set of basic function call from MLIR just for testing purpose
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

void print_cuda_error(hipError_t cuResult) {
  switch (cuResult) {
  case hipSuccess:
    printf("hipSuccess\n");
    break;
  case hipErrorInvalidValue:
    printf("hipErrorInvalidValue\n");
    break;
  case hipErrorOutOfMemory:
    printf("hipErrorOutOfMemory\n");
    break;
  case hipErrorNotInitialized:
    printf("hipErrorNotInitialized\n");
    break;
  case hipErrorDeinitialized:
    printf("hipErrorDeinitialized\n");
    break;
  case hipErrorNoDevice:
    printf("hipErrorNoDevice\n");
    break;
  case hipErrorInvalidDevice:
    printf("hipErrorInvalidDevice\n");
    break;
  case hipErrorInvalidImage:
    printf("hipErrorInvalidImage\n");
    break;
  case hipErrorInvalidContext:
    printf("hipErrorInvalidContext\n");
    break;
  case hipErrorContextAlreadyCurrent:
    printf("hipErrorContextAlreadyCurrent\n");
    break;
  case hipErrorMapFailed:
    printf("hipErrorMapFailed\n");
    break;
  case hipErrorUnmapFailed:
    printf("hipErrorUnmapFailed\n");
    break;
  case hipErrorArrayIsMapped:
    printf("hipErrorArrayIsMapped\n");
    break;
  case hipErrorAlreadyMapped:
    printf("hipErrorAlreadyMapped\n");
    break;
  case hipErrorNoBinaryForGpu:
    printf("hipErrorNoBinaryForGpu\n");
    break;
  case hipErrorAlreadyAcquired:
    printf("hipErrorAlreadyAcquired\n");
    break;
  case hipErrorNotMapped:
    printf("hipErrorNotMapped\n");
    break;
  case hipErrorInvalidSource:
    printf("hipErrorInvalidSource\n");
    break;
  case hipErrorFileNotFound:
    printf("hipErrorFileNotFound\n");
    break;
  case hipErrorInvalidHandle:
    printf("hipErrorInvalidHandle\n");
    break;
  case hipErrorNotFound:
    printf("hipErrorNotFound\n");
    break;
  case hipErrorNotReady:
    printf("hipErrorNotReady\n");
    break;
  case hipErrorLaunchFailure:
    printf("hipErrorLaunchFailure\n");
    break;
  case hipErrorLaunchOutOfResources:
    printf("hipErrorLaunchOutOfResources\n");
    break;
  case hipErrorLaunchTimeOut:
    printf("hipErrorLaunchTimeOut\n");
    break;
  case CUDA_ERROR_LAUNCH_INCOMPATIBLE_TEXTURING:
    printf("CUDA_ERROR_LAUNCH_INCOMPATIBLE_TEXTURING\n");
    break;
  case hipErrorUnknown:
    printf("hipErrorUnknown\n");
    break;
  default:
    printf("Unknown error code\n");
    break;
  }
}

extern "C" int oaru_get_num_devices() {
  int deviceCount = 0;
  hipError_t cuResult = hipGetDeviceCount(&deviceCount);
  if (cuResult != hipSuccess) {
    print_cuda_error(cuResult);
    return 0;
  }
  return deviceCount;
}

extern "C" void oaru_init() {
  hipError_t cuResult = hipInit(0);
  if (cuResult != hipSuccess) {
    print_cuda_error(cuResult);
  }
  hipDevice_t device_;
  cuResult = hipDeviceGet(&device_, 0);
  if (cuResult != hipSuccess) {
    print_cuda_error(cuResult);
    exit(1);
  }

  hipCtx_t context_;
  cuResult = hipCtxCreate(&context_, 0, device_);
  if (cuResult != hipSuccess) {
    print_cuda_error(cuResult);
    hipCtxDestroy(context_);
    exit(1);
  }
}

template <typename T, int N> struct MemRefType {
  T *basePtr;
  T *data;
  int64_t offset;
  int64_t sizes[N];
  int64_t strides[N];
};

extern "C" void* oaru_allocate(void* hostPtr, size_t size) {
  void *devPtr;
  hipError_t cuResult = hipSuccess;
  hipDeviceptr_t cuPtr;
  cuResult = hipMalloc(&cuPtr, size);
  if (cuResult == hipSuccess) {
    devPtr = (void *)(uintptr_t)cuPtr;
    return devPtr;
  } else {
    print_cuda_error(cuResult);
    exit(1);
  }
  return NULL;
}

extern "C" void oaru_print_i32(int val) {
  printf("%d\n", val);
}

template<typename T, int N>
MemRefType<T, N> oaru_allocate_memref(const MemRefType<T, N> *arg) {
  T* devicePtr = (T*)oaru_allocate(arg->basePtr, arg->sizes[0] * sizeof(T));
  struct MemRefType<T, N> allocated;  
  allocated.basePtr = devicePtr;
  allocated.data = devicePtr;
  allocated.offset = arg->offset;
  allocated.sizes[0] = arg->sizes[0];
  allocated.strides[0] = arg->strides[0];
  return allocated;
}

extern "C" MemRefType<float, 1>
oaru_allocate_memref_1d_float(const MemRefType<float, 1> *arg) {
  return oaru_allocate_memref(arg);
}

template<typename T, int N> 
void oaru_free(const MemRefType<T, N> *arg) {
  hipError_t cuResult = hipSuccess;
  hipDeviceptr_t dptr;
  dptr = (hipDeviceptr_t) (uintptr_t) arg->basePtr;
  cuResult = hipFree(dptr);
  if (cuResult != hipSuccess) {
    print_cuda_error(cuResult);
  }
}

extern "C" void
oaru_free_memref_1d_float(const MemRefType<float, 1> *arg) {
  oaru_free(arg);
}

template<typename T, int N1, int N2>
void oaru_update_device(const MemRefType<T, N1> *host, 
    const MemRefType<T, N2> *device) 
{
  hipError_t cuResult = hipSuccess;
  hipDeviceptr_t dptr;
  dptr = (hipDeviceptr_t) (uintptr_t) device->basePtr;
  cuResult = hipMemcpyHtoD(dptr, host->basePtr, count(host) * sizeof(T));
  if (cuResult != hipSuccess) {
    print_cuda_error(cuResult);
    exit(1);
  }
}

extern "C" void
oaru_update_device_1d_float(const MemRefType<float, 1> *host, 
    const MemRefType<float, 1> *device) 
{
  oaru_update_device(host, device);
}

template<typename T, int N1, int N2>
void oaru_update_host(const MemRefType<T, N1> *host, 
    const MemRefType<T, N2> *device) 
{
  hipError_t cuResult = hipSuccess;
  hipDeviceptr_t dptr;
  dptr = (hipDeviceptr_t) (uintptr_t) device->basePtr;
  cuResult = hipMemcpyDtoH(host->basePtr, dptr, count(host) * sizeof(T));
  if (cuResult != hipSuccess) {
    print_cuda_error(cuResult);
    exit(1);
  }
}
template<typename T, int N> 
int64_t count(const MemRefType<T, N> *arg) {
  int count = arg->sizes[0];
  for(int i = 1; i < N; ++i) {
    count *= arg->sizes[i];
  }
  return count;
}

extern "C" void
oaru_update_host_1d_float(const MemRefType<float, 1> *host, 
    const MemRefType<float, 1> *device) 
{
  oaru_update_host(host, device);
}