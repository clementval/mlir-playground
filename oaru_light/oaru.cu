/*
 * Set of basic function call from MLIR just for testing purpose
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

void print_cuda_error(hipError_t cuResult) {
  switch (cuResult) {
  case hipSuccess:
    printf("hipSuccess\n");
    break;
  case hipErrorInvalidValue:
    printf("hipErrorInvalidValue\n");
    break;
  case hipErrorOutOfMemory:
    printf("hipErrorOutOfMemory\n");
    break;
  case hipErrorNotInitialized:
    printf("hipErrorNotInitialized\n");
    break;
  case hipErrorDeinitialized:
    printf("hipErrorDeinitialized\n");
    break;
  case hipErrorNoDevice:
    printf("hipErrorNoDevice\n");
    break;
  case hipErrorInvalidDevice:
    printf("hipErrorInvalidDevice\n");
    break;
  case hipErrorInvalidImage:
    printf("hipErrorInvalidImage\n");
    break;
  case hipErrorInvalidContext:
    printf("hipErrorInvalidContext\n");
    break;
  case hipErrorContextAlreadyCurrent:
    printf("hipErrorContextAlreadyCurrent\n");
    break;
  case hipErrorMapFailed:
    printf("hipErrorMapFailed\n");
    break;
  case hipErrorUnmapFailed:
    printf("hipErrorUnmapFailed\n");
    break;
  case hipErrorArrayIsMapped:
    printf("hipErrorArrayIsMapped\n");
    break;
  case hipErrorAlreadyMapped:
    printf("hipErrorAlreadyMapped\n");
    break;
  case hipErrorNoBinaryForGpu:
    printf("hipErrorNoBinaryForGpu\n");
    break;
  case hipErrorAlreadyAcquired:
    printf("hipErrorAlreadyAcquired\n");
    break;
  case hipErrorNotMapped:
    printf("hipErrorNotMapped\n");
    break;
  case hipErrorInvalidSource:
    printf("hipErrorInvalidSource\n");
    break;
  case hipErrorFileNotFound:
    printf("hipErrorFileNotFound\n");
    break;
  case hipErrorInvalidHandle:
    printf("hipErrorInvalidHandle\n");
    break;
  case hipErrorNotFound:
    printf("hipErrorNotFound\n");
    break;
  case hipErrorNotReady:
    printf("hipErrorNotReady\n");
    break;
  case hipErrorLaunchFailure:
    printf("hipErrorLaunchFailure\n");
    break;
  case hipErrorLaunchOutOfResources:
    printf("hipErrorLaunchOutOfResources\n");
    break;
  case hipErrorLaunchTimeOut:
    printf("hipErrorLaunchTimeOut\n");
    break;
  case CUDA_ERROR_LAUNCH_INCOMPATIBLE_TEXTURING:
    printf("CUDA_ERROR_LAUNCH_INCOMPATIBLE_TEXTURING\n");
    break;
  case hipErrorUnknown:
    printf("hipErrorUnknown\n");
    break;
  default:
    printf("Unknown error code\n");
    break;
  }
}

extern "C" int oaru_get_num_devices() {
  int deviceCount = 0;
  hipError_t cuResult = hipGetDeviceCount(&deviceCount);
  if (cuResult != hipSuccess) {
    print_cuda_error(cuResult);
    return 0;
  }
  return deviceCount;
}

extern "C" void oaru_init() {
  hipError_t cuResult = hipInit(0);
  if (cuResult != hipSuccess) {
    print_cuda_error(cuResult);
  }
  hipDevice_t device_;
  cuResult = hipDeviceGet(&device_, 0);
  if (cuResult != hipSuccess) {
    print_cuda_error(cuResult);
    exit(1);
  }

  hipCtx_t context_;
  cuResult = hipCtxCreate(&context_, 0, device_);
  if (cuResult != hipSuccess) {
    print_cuda_error(cuResult);
    hipCtxDestroy(context_);
    exit(1);
  }
}

template <typename T, int N> struct MemRefType {
  T *basePtr;
  T *data;
  int64_t offset;
  int64_t sizes[N];
  int64_t strides[N];
};

extern "C" void* oaru_allocate(void* hostPtr, size_t size) {
  void *devPtr;
  hipError_t cuResult = hipSuccess;
  hipDeviceptr_t cuPtr;
  cuResult = hipMalloc(&cuPtr, size);
  if (cuResult == hipSuccess) {
    devPtr = (void *)(uintptr_t)cuPtr;
    return devPtr;
  } else {
    print_cuda_error(cuResult);
    exit(1);
  }
  return NULL;
}

extern "C" void oaru_print_i32(int val) {
  printf("%d\n", val);
}

template<typename T, int N>
MemRefType<T, N> oaru_allocate_memref(const MemRefType<T, N> *arg) {
  T* devicePtr = (T*)oaru_allocate(arg->basePtr, count(arg) * sizeof(T));
  
  struct MemRefType<T, N> allocated;  
  allocated.basePtr = devicePtr;
  allocated.data = devicePtr;
  allocated.offset = arg->offset;
  allocated.sizes[0] = arg->sizes[0];
  allocated.strides[0] = arg->strides[0];
  return allocated;
}

extern "C" MemRefType<float, 1>
oaru_allocate_memref_1d_float(float *allocated,
                              float *aligned, int64_t offset,
                              int64_t size, int64_t stride) {
  MemRefType<float, 1> descriptor;
  descriptor.basePtr = allocated;
  descriptor.data = aligned;
  descriptor.offset = offset;
  descriptor.sizes[0] = size;
  descriptor.strides[0] = stride;
  return oaru_allocate_memref(&descriptor);
}

template<typename T, int N> 
void oaru_free(const MemRefType<T, N> *arg) {
  hipError_t cuResult = hipSuccess;
  hipDeviceptr_t dptr;
  dptr = (hipDeviceptr_t) (uintptr_t) arg->basePtr;
  cuResult = hipFree(dptr);
  if (cuResult != hipSuccess) {
    print_cuda_error(cuResult);
  }
}

extern "C" void
oaru_free_memref_1d_float(float *allocated,
                          float *aligned, int64_t offset,
                          int64_t size, int64_t stride) {
  MemRefType<float, 1> descriptor;
  descriptor.basePtr = allocated;
  descriptor.data = aligned;
  descriptor.offset = offset;
  descriptor.sizes[0] = size;
  descriptor.strides[0] = stride;
  oaru_free(&descriptor);
}

template<typename T, int N>
void oaru_update_device(const MemRefType<T, N> *host, 
                        const MemRefType<T, N> *device) 
{
  hipError_t cuResult = hipSuccess;
  hipDeviceptr_t dptr;
  dptr = (hipDeviceptr_t) (uintptr_t) device->basePtr;
  cuResult = hipMemcpyHtoD(dptr, host->basePtr, count(host) * sizeof(T));
  if (cuResult != hipSuccess) {
    print_cuda_error(cuResult);
    exit(1);
  }
}

extern "C" void
oaru_update_device_1d_float(float *host_allocated, float *host_aligned, 
                            int64_t host_offset, int64_t host_size, 
                            int64_t host_stride, 
                            float *device_allocated, float *device_aligned, 
                            int64_t device_offset, int64_t device_size, 
                            int64_t device_stride) 
{
  MemRefType<float, 1> host_descriptor;
  host_descriptor.basePtr = host_allocated;
  host_descriptor.data = host_aligned;
  host_descriptor.offset = host_offset;
  host_descriptor.sizes[0] = host_size;
  host_descriptor.strides[0] = host_stride;

  MemRefType<float, 1> device_descriptor;
  device_descriptor.basePtr = device_allocated;
  device_descriptor.data = device_aligned;
  device_descriptor.offset = device_offset;
  device_descriptor.sizes[0] = device_size;
  device_descriptor.strides[0] = device_stride;
  oaru_update_device(&host_descriptor, &device_descriptor);
}

template<typename T, int N1, int N2>
void oaru_update_host(const MemRefType<T, N1> *host, 
                      const MemRefType<T, N2> *device)
{
  hipError_t cuResult = hipSuccess;
  hipDeviceptr_t dptr;
  dptr = (hipDeviceptr_t) (uintptr_t) device->basePtr;
  cuResult = hipMemcpyDtoH(host->basePtr, dptr, count(host) * sizeof(T));
  if (cuResult != hipSuccess) {
    print_cuda_error(cuResult);
    exit(1);
  }
}

template<typename T, int N> 
int64_t count(const MemRefType<T, N> *arg) {
  int count = arg->sizes[0];
  for(int i = 1; i < N; ++i) {
    count *= arg->sizes[i];
  }
  return count;
}

extern "C" void
oaru_update_host_1d_float(float *host_allocated, float *host_aligned, 
                          int64_t host_offset, int64_t host_size, 
                          int64_t host_stride, 
                          float *device_allocated, float *device_aligned, 
                          int64_t device_offset, int64_t device_size, 
                          int64_t device_stride) 
{
  MemRefType<float, 1> host_descriptor;
  host_descriptor.basePtr = host_allocated;
  host_descriptor.data = host_aligned;
  host_descriptor.offset = host_offset;
  host_descriptor.sizes[0] = host_size;
  host_descriptor.strides[0] = host_stride;

  MemRefType<float, 1> device_descriptor;
  device_descriptor.basePtr = device_allocated;
  device_descriptor.data = device_aligned;
  device_descriptor.offset = device_offset;
  device_descriptor.sizes[0] = device_size;
  device_descriptor.strides[0] = device_stride;

  oaru_update_host(&host_descriptor, &device_descriptor);
}