#include <hip/hip_runtime.h>
#include <stdio.h>

extern "C"int oaru_get_num_devices() {
  int deviceCount = 0;
  hipError_t cuResult = hipGetDeviceCount(&deviceCount);
  if (cuResult != hipSuccess) {
    printf("[ERROR] Cannot read number of devices\n");
  }
  return deviceCount;
}

