#include <hip/hip_runtime.h>
#include <stdio.h>

void print_cuda_error(hipError_t cuResult) {
  switch (cuResult) {
  case hipSuccess:
    printf("CUDA_SUCCESS\n");
    break;
  case hipErrorInvalidValue:
    printf("CUDA_ERROR_INVALID_VALUE\n");
    break;
  case hipErrorOutOfMemory:
    printf("CUDA_ERROR_OUT_OF_MEMORY\n");
    break;
  case hipErrorNotInitialized:
    printf("CUDA_ERROR_NOT_INITIALIZED\n");
    break;
  case hipErrorDeinitialized:
    printf("CUDA_ERROR_DEINITIALIZED\n");
    break;
  case hipErrorNoDevice:
    printf("CUDA_ERROR_NO_DEVICE\n");
    break;
  case hipErrorInvalidDevice:
    printf("CUDA_ERROR_INVALID_DEVICE\n");
    break;
  case hipErrorInvalidImage:
    printf("CUDA_ERROR_INVALID_IMAGE\n");
    break;
  case hipErrorInvalidContext:
    printf("CUDA_ERROR_INVALID_CONTEXT\n");
    break;
  case hipErrorContextAlreadyCurrent:
    printf("CUDA_ERROR_CONTEXT_ALREADY_CURRENT\n");
    break;
  case hipErrorMapFailed:
    printf("CUDA_ERROR_MAP_FAILED\n");
    break;
  case hipErrorUnmapFailed:
    printf("CUDA_ERROR_UNMAP_FAILED\n");
    break;
  case hipErrorArrayIsMapped:
    printf("CUDA_ERROR_ARRAY_IS_MAPPED\n");
    break;
  case hipErrorAlreadyMapped:
    printf("CUDA_ERROR_ALREADY_MAPPED\n");
    break;
  case hipErrorNoBinaryForGpu:
    printf("CUDA_ERROR_NO_BINARY_FOR_GPU\n");
    break;
  case hipErrorAlreadyAcquired:
    printf("CUDA_ERROR_ALREADY_ACQUIRED\n");
    break;
  case hipErrorNotMapped:
    printf("CUDA_ERROR_NOT_MAPPED\n");
    break;
  case hipErrorInvalidSource:
    printf("CUDA_ERROR_INVALID_SOURCE\n");
    break;
  case hipErrorFileNotFound:
    printf("CUDA_ERROR_FILE_NOT_FOUND\n");
    break;
  case hipErrorInvalidHandle:
    printf("CUDA_ERROR_INVALID_HANDLE\n");
    break;
  case hipErrorNotFound:
    printf("CUDA_ERROR_NOT_FOUND\n");
    break;
  case hipErrorNotReady:
    printf("CUDA_ERROR_NOT_READY\n");
    break;
  case hipErrorLaunchFailure:
    printf("CUDA_ERROR_LAUNCH_FAILED\n");
    break;
  case hipErrorLaunchOutOfResources:
    printf("CUDA_ERROR_LAUNCH_OUT_OF_RESOURCES\n");
    break;
  case hipErrorLaunchTimeOut:
    printf("CUDA_ERROR_LAUNCH_TIMEOUT\n");
    break;
  case CUDA_ERROR_LAUNCH_INCOMPATIBLE_TEXTURING:
    printf("CUDA_ERROR_LAUNCH_INCOMPATIBLE_TEXTURING\n");
    break;
  case hipErrorUnknown:
    printf("CUDA_ERROR_UNKNOWN\n");
    break;
  default:
    printf("Unknown error code\n");
    break;
  }
}

extern "C" int oaru_get_num_devices() {
  int deviceCount = 0;
  hipError_t cuResult = hipGetDeviceCount(&deviceCount);
  if (cuResult != hipSuccess) {
    print_cuda_error(cuResult);
    return 0;
  }
  return deviceCount;
}

extern "C" void oaru_init() {
  hipError_t cuResult = hipInit(0);
  if (cuResult != hipSuccess) {
    print_cuda_error(cuResult);
  }
}

extern "C" void oaru_print_i32(int val) {
  printf("%d\n", val);
}

